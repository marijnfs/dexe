#include "hip/hip_runtime.h"
#include "gate.h"

__global__ void gate_kerneld(int N, double const *a, double const *b, double *out) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	out[i] += a[i] * b[i];
}

__global__ void gate_kernelf(int N, float const *a, float const *b, float *out) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	out[i] += a[i] * b[i];
}

template <>
void gate<double>(Tensor<double> &a, Tensor<double> &b, Tensor<double> &out) {
	int s = a.size();
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (s + BLOCKSIZE - 1) / BLOCKSIZE );

	gate_kerneld<<<dimGrid, dimBlock>>>(s, a.data, b.data, out.data);
}

template <>
void gate<float>(Tensor<float> &a, Tensor<float> &b, Tensor<float> &out) {
	int s = a.size();
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (s  + BLOCKSIZE - 1) / BLOCKSIZE);

	gate_kernelf<<<dimGrid, dimBlock>>>(s, a.data, b.data, out.data);
}

///range

__global__ void range_kerneld(double *a, int N, double const min, double const max) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	a[i] = a[i] * (max - min) + min;
}

__global__ void range_kernelf(float *a, int N, float const min, float const max) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	a[i] = 	a[i] * (max - min) + min;
}


template <>
void range<float>(float *a, int N, float const min, float const max) {
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (N + BLOCKSIZE - 1) / BLOCKSIZE );

	range_kernelf<<<dimGrid, dimBlock>>>(a, N, min, max);
}

template <>
void range<double>(double *a, int N, double const min, double const max) {
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (N + BLOCKSIZE - 1) / BLOCKSIZE );

	range_kerneld<<<dimGrid, dimBlock>>>(a, N, min, max);
}

__global__ void tanh_deriv_kernelf(float *out_err, float *act, float *in_err, int N) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	in_err[i] = (1.0 - (act[i] * act[i])) * out_err[i];
}

__global__ void tanh_deriv_kerneld(double *out_err, double *act, double *in_err, int N) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	in_err[i] = (1.0 - (act[i] * act[i])) * out_err[i];
}

template <>
void tanh_deriv<float>(float *out_err, float *act, float *in_err, int n) {
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (n  + BLOCKSIZE - 1) / BLOCKSIZE);

	tanh_deriv_kernelf<<<dimGrid, dimBlock>>>(out_err, act, in_err, n);
}

template <>
void tanh_deriv<double>(double *out_err, double *act, double *in_err, int n) {
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (n  + BLOCKSIZE - 1) / BLOCKSIZE);

	tanh_deriv_kerneld<<<dimGrid, dimBlock>>>(out_err, act, in_err, n);
}

__global__ void sigm_deriv_kernelf(float *out_err, float *act, float *in_err, int N) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	in_err[i] = (1.0 - act[i]) * act[i] * out_err[i];
}

__global__ void sigm_deriv_kerneld(double *out_err, double *act, double *in_err, int N) {
	int const i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	in_err[i] = (1.0 - act[i]) * act[i] * out_err[i];
}

template <>
void sigm_deriv<float>(float *out_err, float *act, float *in_err, int n) {
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (n  + BLOCKSIZE - 1) / BLOCKSIZE);

	sigm_deriv_kernelf<<<dimGrid, dimBlock>>>(out_err, act, in_err, n);
}

template <>
void sigm_deriv<double>(double *out_err, double *act, double *in_err, int n) {
	int const BLOCKSIZE(1024);

	int dimBlock( BLOCKSIZE );
	int dimGrid( (n  + BLOCKSIZE - 1) / BLOCKSIZE);

	sigm_deriv_kerneld<<<dimGrid, dimBlock>>>(out_err, act, in_err, n);
}
