#include "hip/hip_runtime.h"
#include "util.h"
#include "handler.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__global__ void normal_kernel(int seed, float *data, int n, float mean, float std) {
  if (threadIdx.x != 0) return;
  hiprandState state;
  
  hiprand_init(seed, 0, 0, &state);
  for (size_t i(0); i < n; ++i)
    data[i] = hiprand_normal(&state) * std + mean;
}

__global__ void normal_kerneld(int seed, double *data, int n, double mean, double std) {
  if (threadIdx.x != 0) return;
  hiprandState state;
  hiprand_init(seed, 0, 0, &state);
  for (size_t i(0); i < n; ++i)
    data[i] = hiprand_normal_double(&state) * std + mean;
}

template <>
void init_normal<float>(float *a, int N, float mean, float std) {
     normal_kernel<<<1, 32>>>(rand(), a, N, mean, std);
}

template <>
void init_normal<double>(double *a, int N, double mean, double std) {
     normal_kerneld<<<1, 32>>>(rand(), a, N, mean, std);
}